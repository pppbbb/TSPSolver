#include "hip/hip_runtime.h"
#include <cstdio>

extern "C"
{
    __global__
    void CreateCreatures(int* creatures, int creatures_no, int creature_size)
    {
        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);
        if(pos < creatures_no * creature_size)
            creatures[pos] = (pos % creature_size);
    }

    __global__
    void BitonicSortStep(int* creatures, int* ranks, int j, int k)
    {
        unsigned int i, ixj;
        i = threadIdx.x + blockDim.x * blockIdx.x;
        ixj = i^j;

        if ((ixj) > i)
        {
            if((i & k) == 0)
                if(ranks[i] > ranks[ixj]) // swap
                {
                    int temp = ranks[i];
                    ranks[i] = ranks[ixj];
                    ranks[ixj] = temp;

                    temp = creatures[i];
                    creatures[i] = creatures[ixj];
                    creatures[ixj] = temp;
                }

            if((i&k)!=0)
                if(ranks[i]<ranks[ixj]) // swap
                {
                    int temp = ranks[i];
                    ranks[i] = ranks[ixj];
                    ranks[ixj] = temp;

                    temp = creatures[i];
                    creatures[i] = creatures[ixj];
                    creatures[ixj] = temp;
                }
          }
    }

    __global__
    void WhichReverse(int* array, int permutation_size, int permutation_no, int* to_reverse)
    {
        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);
        if(pos < permutation_size * permutation_no)
        {
            if(pos % permutation_size != 0)
            {
                if(array[pos - 1] > array[pos])
                {
                    to_reverse[pos / permutation_size] = 1;
                }
            }
        }
    }


    __global__
    void BitonicToAscendingOrder(int* array, int* ranks, int* to_reverse, int permutation_size, int permutation_no)
    {
        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);

        if(pos < permutation_no * permutation_size && to_reverse[pos/permutation_size] == 1)
        {
            if(pos % permutation_size < (permutation_size / 2))
            {
                int second_pos = pos - (pos% permutation_size) * 2 + permutation_size - 1;

                int tmp = array[second_pos];
                array[second_pos] = array[pos];
                array[pos] = tmp;

                tmp = ranks[second_pos];
                ranks[second_pos] = ranks[pos];
                ranks[pos] = tmp;
            }
        }
    }

    __global__
    void SetRandomRanks(int seed, int* ranks, int size)
    {
        int thid = ((blockIdx.x * blockDim.x) + threadIdx.x ) + 1;
        if(thid - 1 < size)
            ranks[thid - 1] = (  (((( (long long)seed * thid) % 24837) + ((long long)seed * thid % 21447))) % 21474)% 21474;
    }

    __global__
    void SetMutationRanks(int* ranks, int seed, int probability, int rank_difference, int permutation_size, int permutation_no)
    {
        int thid = ((blockIdx.x * blockDim.x) + threadIdx.x ) + 1;

        if(thid - 1 < permutation_size * permutation_no)
        {
            int pro_rand = (( (long long) seed * thid ) % 2000001557) *  ( ((long long) seed * thid) % 2000001557) % 1000000000;

            if(pro_rand < probability)
               ranks[thid - 1] = (  (((( (long long)seed * thid) % 2147483647) + ((long long)seed * thid % 2147483647)) * (long long)seed ) % 2147483647)% 2147483647;
            else
                ranks[thid - 1] = ((thid - 1) % permutation_size) * rank_difference;
        }
    }

    __global__
    void SetValue(int* array, int size, int value)
    {
        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);
        if(pos < size)
            array[pos] = value;
    }

    __global__
    void AssignHalfPopulation(int* crossover_array, int* population, int crossover_no, int creature_size, int begin_point, int* left)
    {
        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);
        if((pos < crossover_no * crossover_no * creature_size) && ((pos % creature_size) < creature_size / 2))
        {
            population[pos] = crossover_array[(pos / (creature_size * crossover_no)) * creature_size + (pos % creature_size) + begin_point];
            left[pos - (pos % creature_size) + population[pos]] = 0;
        }
    }

    __global__
    void FindMissingGens(int* crossover_array, int* left, int* co2_ind, int crossover_no, int creature_size)
    {
        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);

        if(pos < crossover_no * crossover_no * creature_size)
        {
            if(left[pos - (pos % creature_size) + crossover_array[pos % (crossover_no * creature_size)]] == 1)
                co2_ind[pos] = 1;
            else
                co2_ind[pos] = 0;
        }
    }

    __global__
    void FillRestPopulation(int* crossover_array, int* population, int* prefix_sum, int crossover_no, int creature_size, int begin_point)
    {
        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);
        int to_fill = pos - (pos % creature_size) + begin_point;
        if(pos < crossover_no * crossover_no * creature_size)
        {
            if(pos % creature_size == 0)
            {
                if(prefix_sum[pos] == 1)
                    population[to_fill + 1] = crossover_array[pos % (crossover_no * creature_size)];

            }
            else
            {
                if(prefix_sum[pos - 1] < prefix_sum[pos])
                    population[to_fill + prefix_sum[pos]] = crossover_array[pos % (crossover_no * creature_size)];
            }

        }
    }

    __global__
    void PrefixSumPrecalculate(int* array, int* blocks_sum, int permutation_size, int blocks_per_permutation)
    {
        int my_permutation = blockIdx.x / blocks_per_permutation;
        int perm_pos = ((blockIdx.x % blocks_per_permutation) * blockDim.x) + threadIdx.x;
        int a_pos = (my_permutation * permutation_size) + perm_pos;

        __shared__ int sum[1024];

        if(perm_pos < permutation_size)
            sum[threadIdx.x] = array[a_pos];
        else
            sum[threadIdx.x] = 0;

        int tmp_sum = 0;

        for(int i = 1; i < 1024; i*= 2)
        {
            syncthreads();

            if(threadIdx.x >= i)
                tmp_sum = sum[threadIdx.x - i];
            else
                tmp_sum = 0;

            syncthreads();

            sum[threadIdx.x] += tmp_sum;
        }

        syncthreads();

        if(threadIdx.x == 0)
            blocks_sum[blockIdx.x] = sum[1023];
    }

    __global__
    void CalculateBlocksPrefixSum(int* blocks_sum, int* addition, int blocks_per_permutation)
    {
        if(threadIdx.x == 0)
        {
            int end = (blockIdx.x + 1) * blocks_per_permutation;
            addition[blockIdx.x * blocks_per_permutation] = 0;

            for(int i = (blockIdx.x * blocks_per_permutation) + 1; i < end; ++i)
                addition[i] = addition[i - 1] + blocks_sum[i - 1];
        }
    }

    __global__
    void CalculateFinalPrefixSum(int* array, int* addition, int permutation_size, int blocks_per_permutation)
    {
        int my_permutation = blockIdx.x / blocks_per_permutation;
        int perm_pos = ((blockIdx.x % blocks_per_permutation) * blockDim.x) + threadIdx.x;
        int a_pos = (my_permutation * permutation_size) + perm_pos;

        __shared__ int sum[1024];

        if(perm_pos < permutation_size)
            sum[threadIdx.x] = array[a_pos];
        else
            sum[threadIdx.x] = 0;

        if(threadIdx.x == 0)
            sum[0] += addition[blockIdx.x];

        int tmp_sum = 0;

        for(int i = 1; i < 1024; i*= 2)
        {
            syncthreads();

            if(threadIdx.x >= i)
                tmp_sum = sum[threadIdx.x - i];
            else
                tmp_sum = 0;

            syncthreads();

            sum[threadIdx.x] += tmp_sum;
        }

        syncthreads();

        if(perm_pos < permutation_size)
            array[a_pos] = sum[perm_pos];
    }

     __global__
    void SetPathWeight(int* population, int* weight, int* graph, int creatures_no, int creature_size)
    {
        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);

        if(pos < creature_size * creatures_no)
        {
            int from = population[pos];
            int to = population[pos + 1 - (((pos % creature_size) == (creature_size - 1))? creature_size: 0)];
            weight[pos] = graph[from * creature_size + to];
        }
    }

    __global__
    void FillRanksFromDistance(int* distance, int* ranks, int creatures_no, int creature_size)
    {
        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);
        if(pos < creatures_no)
            ranks[pos] = distance[((pos + 1) * creature_size) - 1];
    }

    __global__
    void SetWhoBecomeMaster(int* results, int* population_scores, int* bests_scores, int bests_size) {

        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);

        if(pos < bests_size)
            if(population_scores[pos] < bests_scores[bests_size - 1 - pos])
                results[pos] = 1;
            else
                results[pos] = 0;
    }

    __global__
    void AddNewMasters(int new_masters_no, int* population, int* masters, int* population_scores, int* population_ranking,
                       int* bests_scores, int bests_size, int individual_size)
    {

        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);
        if(pos <(new_masters_no  * individual_size ))
        {
            int which = pos/individual_size;
            int index = population_ranking[which];
            int elem = pos % individual_size;
            int dest_index = bests_size - new_masters_no + which;
            masters[dest_index * individual_size + elem] = population[index * individual_size + elem];
            bests_scores[dest_index] = population_scores[which];
        }
    }

    __global__
    void CopyFromTo(int* from, int from_start_index, int* to, int to_start_index, int permutation_no, int permutation_size){

        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);
        if(pos < (permutation_no * permutation_size))
        {
            to[pos + to_start_index] = from[pos + from_start_index];
        }
    }

    __global__
    void CopyLocalBestsToCrossover(int* population, int* population_ranking, int best_population_size, int new_bests_no, int* crossover, int bests_size, int individual_size)
    {

        int pos = ((blockIdx.x * blockDim.x) + threadIdx.x);
        if(pos < (best_population_size  * individual_size ))
        {
            int which = pos/individual_size + new_bests_no;
            int index = population_ranking[which];
            int elem = pos % individual_size;
            int dest_index = bests_size + (pos/individual_size);

            crossover[dest_index * individual_size + elem] = population[index * individual_size + elem];
        }
    }

    __global__
    void GetValue(int* from, int* to, int at)
    {
        if(threadIdx.x == 0)
            to[0] = from[at];
    }
}
